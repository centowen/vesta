#include "hip/hip_runtime.h"
// vesta, use Ceres to douv-model fitting for ms-data.
// Copyright (C) 2015  Lukas Lindroos
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public License
// as published by the Free Software Foundation; either version 2
// of the License, or (at your option) any later version.
// 
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. 
//
#include "CommonCuda.h"
#include "cuda_error.h"
#include "Chunk.h"

const float C_LIGHT = 299792458;
__global__ void sqrt_weights(float* weights, const int nchan, const int nstokes, const int nrow);
__global__ void cu_gaussian_size(double sigma, double x0, double y0,
                                 const int nchan, const int nstokes, const int nrow,
                                 double* u, double* v,
                                 double* size);
__global__ void cu_evaluate_gaussian(float flux, float sigma, float x0, float y0,
                             const int nchan, const int nstokes, const int nrow,
					         const VisDataContainer uvdata,
							 float* residuals, float* jacobians);
__global__ void cu_disk_size(double sigma, double x0, double y0,
                             const int nchan, const int nstokes, const int nrow,
                             double* u, double* v,
                             double* size, double* dsize_dsigma);
__global__ void cu_pos(double sigma, double x0, double y0,
                       const int nchan, const int nstokes, const int nrow,
                       double* u, double* v,
                       double* pos_real, double* pos_imag);

void allocate_stuff(const int nchan, const int nstokes, const int nrow, DataContainer& data)
{
	CudaSafeCall(hipMalloc( (void**)&data.u, sizeof(double)*nchan*nstokes*nrow));
	CudaSafeCall(hipMalloc( (void**)&data.v, sizeof(double)*nchan*nstokes*nrow));
	CudaSafeCall(hipMalloc( (void**)&data.size, sizeof(double)*nchan*nstokes*nrow));
	CudaSafeCall(hipMalloc( (void**)&data.dsize_dsigma, sizeof(double)*nchan*nstokes*nrow));
	CudaSafeCall(hipMalloc( (void**)&data.pos_real, sizeof(double)*nchan*nstokes*nrow));
	CudaSafeCall(hipMalloc( (void**)&data.pos_imag, sizeof(double)*nchan*nstokes*nrow));
}


// Allocate space for uv data on device and copy data over.
void setup_uvdata(Chunk& chunk, VisDataContainer& dev_uvdata)
{
	dev_uvdata.nchan = chunk.nChan();
	dev_uvdata.nstokes = chunk.nStokes();
	dev_uvdata.nrow = chunk.size();


	size_t size = sizeof(float)*dev_uvdata.nrow*dev_uvdata.nstokes*dev_uvdata.nchan;
	CudaSafeCall(hipMalloc((void**)&dev_uvdata.u, size));
	CudaSafeCall(hipMalloc((void**)&dev_uvdata.v, size));
	CudaSafeCall(hipMalloc((void**)&dev_uvdata.w, size));
	CudaSafeCall(hipMalloc((void**)&dev_uvdata.sqrt_weights, size));
	CudaSafeCall(hipMalloc((void**)&dev_uvdata.V_real, size));
	CudaSafeCall(hipMalloc((void**)&dev_uvdata.V_imag, size));

	// Copy u, v and w to device.
	float* u = new float[dev_uvdata.nrow*dev_uvdata.nstokes*dev_uvdata.nchan];
	float* v = new float[dev_uvdata.nrow*dev_uvdata.nstokes*dev_uvdata.nchan];
	float* w = new float[dev_uvdata.nrow*dev_uvdata.nstokes*dev_uvdata.nchan];
	for(int uvrow = 0; uvrow < chunk.size(); uvrow++)
	{
		Visibility& inVis = chunk.inVis[uvrow];
		float* freq = inVis.freq;

		for(int chan = 0; chan < chunk.nChan(); chan++)
		{
			for(int pol = 0; pol < chunk.nStokes(); pol++)
			{
				size_t index = uvrow*chunk.nChan()*chunk.nStokes()+pol*chunk.nChan()+chan;
				u[index] = inVis.u * freq[chan] / C_LIGHT;
				v[index] = inVis.v * freq[chan] / C_LIGHT;
				w[index] = inVis.w * freq[chan] / C_LIGHT;
			}
		}
	}
	CudaSafeCall(hipMemcpy(dev_uvdata.u, u, size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(dev_uvdata.v, v, size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(dev_uvdata.w, w, size, hipMemcpyHostToDevice));
	delete[] u;
	delete[] v;
	delete[] w;


	CudaSafeCall(hipMemcpy(dev_uvdata.sqrt_weights, chunk.weight_in, size, hipMemcpyHostToDevice));
	dim3 dimBlock(dev_uvdata.nchan, dev_uvdata.nstokes);
	dim3 dimGrid(dev_uvdata.nrow);
// 	sqrt_weights<<<dimGrid, dimBlock>>>(dev_uvdata.sqrt_weights,
// 	                                    dev_uvdata.nchan, dev_uvdata.nstokes, dev_uvdata.nrow);
	CudaSafeCall(hipMemcpy(dev_uvdata.V_real, chunk.data_real_in, size, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(dev_uvdata.V_imag, chunk.data_imag_in, size, hipMemcpyHostToDevice));

	size = sizeof(int)*dev_uvdata.nrow*dev_uvdata.nstokes*dev_uvdata.nchan;
	CudaSafeCall(hipMalloc((void**)&dev_uvdata.flag, size));
	CudaSafeCall(hipMemcpy(dev_uvdata.flag, chunk.data_flag_in, size, hipMemcpyHostToDevice));
}

void free_uvdata(VisDataContainer& dev_uvdata)
{
	CudaSafeCall(hipFree(dev_uvdata.u));
	CudaSafeCall(hipFree(dev_uvdata.v));
	CudaSafeCall(hipFree(dev_uvdata.w));
	CudaSafeCall(hipFree(dev_uvdata.sqrt_weights));
	CudaSafeCall(hipFree(dev_uvdata.V_real));
	CudaSafeCall(hipFree(dev_uvdata.V_imag));
	CudaSafeCall(hipFree(dev_uvdata.flag));
}

__global__ void sqrt_weights(float* weights, const int nchan, const int nstokes, const int nrow)
{
	size_t chan = threadIdx.x;
	size_t pol = threadIdx.y;
	size_t row = blockIdx.x;

	if(chan < nchan and pol < nstokes and row < nrow)
	{
		size_t index = chan+pol*nchan+row*nchan*nstokes;
		if(weights[index] != 0)
			weights[index] = sqrt(weights[index]);
	}
}

void free_stuff(DataContainer& data)
{
	CudaSafeCall(hipFree( data.u));
	CudaSafeCall(hipFree( data.v));
	CudaSafeCall(hipFree( data.size));
	CudaSafeCall(hipFree( data.dsize_dsigma));
	CudaSafeCall(hipFree( data.pos_real));
	CudaSafeCall(hipFree( data.pos_imag));
}

void evaluate_gaussian(float flux, float sigma, float x0, float y0,
                       const int nchan, const int nstokes, const int nrow,
					   const VisDataContainer uvdata,
					   float* residuals, float** jacobians)
{
	dim3 dimBlock(uvdata.nchan, uvdata.nstokes);
	dim3 dimGrid(uvdata.nrow);
// 	dim3 dimBlock(nchan, nstokes);
// 	dim3 dimGrid(nrow);
	
	// setup storage for resiudals and jacobians
	float* dev_residuals;
	float* dev_jacobians;

// 	int nres = nchan*nstokes*nrow*2;
	int nres = uvdata.nchan*uvdata.nstokes*uvdata.nrow*2;
	CudaSafeCall(hipMalloc((void**)&dev_residuals, sizeof(float)*nres));
	if(jacobians != NULL)
		CudaSafeCall(hipMalloc((void**)&dev_jacobians, sizeof(float)*nres*4));
	else
		dev_jacobians = NULL;

	cu_evaluate_gaussian<<<dimGrid, dimBlock>>>(flux, sigma, x0, y0,
			                                    uvdata.nchan, uvdata.nstokes, uvdata.nrow,
												uvdata, dev_residuals, dev_jacobians);

	CudaSafeCall(hipMemcpy(residuals, dev_residuals, sizeof(float)*nres, hipMemcpyDeviceToHost));
	if(jacobians != NULL)
	{
		if(jacobians[0] != NULL)
		{
			CudaSafeCall(hipMemcpy(jacobians[0], &dev_jacobians[nres*0],
						sizeof(float)*nres, hipMemcpyDeviceToHost));
		}
		if(jacobians[1] != NULL)
		{
			CudaSafeCall(hipMemcpy(jacobians[1], &dev_jacobians[nres*1],
						sizeof(float)*nres, hipMemcpyDeviceToHost));
		}
		if(jacobians[2] != NULL)
		{
			CudaSafeCall(hipMemcpy(jacobians[2], &dev_jacobians[nres*2],
						sizeof(float)*nres, hipMemcpyDeviceToHost));
		}
		if(jacobians[3] != NULL)
		{
			CudaSafeCall(hipMemcpy(jacobians[3], &dev_jacobians[nres*3],
						sizeof(float)*nres, hipMemcpyDeviceToHost));
		}
	}
	CudaSafeCall(hipFree(dev_residuals));
	if(dev_jacobians != NULL)
		CudaSafeCall(hipFree(dev_jacobians));
}

void calc_gaussian(double sigma, double x0, double y0,
                   const int nchan, const int nstokes, const int nrow,
                   double* u, double* v,
                   double* size, double* pos_real, double* pos_imag,
				   const DataContainer data)
{
	CudaSafeCall(hipMemcpy( data.u, u, sizeof(double)*nchan*nstokes*nrow, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy( data.v, v, sizeof(double)*nchan*nstokes*nrow, hipMemcpyHostToDevice));
	dim3 dimBlock(nchan, nstokes);
	dim3 dimGrid(nrow);
	cu_gaussian_size<<<dimGrid, dimBlock>>>(sigma, x0, y0, nchan, nstokes, nrow,
	                                        data.u, data.v, data.size);
	cu_pos<<<dimGrid, dimBlock>>>(sigma, x0, y0, nchan, nstokes, nrow,
	                              data.u, data.v, data.pos_real, data.pos_imag);

	CudaSafeCall(hipMemcpy(size, data.size, sizeof(double)*nchan*nstokes*nrow, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(pos_real, data.pos_real, sizeof(double)*nchan*nstokes*nrow, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(pos_imag, data.pos_imag, sizeof(double)*nchan*nstokes*nrow, hipMemcpyDeviceToHost));
}


void calc_disk(double sigma, double x0, double y0,
               const int nchan, const int nstokes, const int nrow,
               double* u, double* v,
               double* size, double* dsize_dsigma, double* pos_real, double* pos_imag,
			   const DataContainer data)
{
	CudaSafeCall(hipMemcpy( data.u, u, sizeof(double)*nchan*nstokes*nrow, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy( data.v, v, sizeof(double)*nchan*nstokes*nrow, hipMemcpyHostToDevice));
	dim3 dimBlock(nchan, nstokes);
	dim3 dimGrid(nrow);
	cu_disk_size<<<dimGrid, dimBlock>>>(sigma, x0, y0, nchan, nstokes, nrow,
	                                        data.u, data.v, data.size, data.dsize_dsigma);
	cu_pos<<<dimGrid, dimBlock>>>(sigma, x0, y0, nchan, nstokes, nrow,
	                              data.u, data.v, data.pos_real, data.pos_imag);

	CudaSafeCall(hipMemcpy(size, data.size, sizeof(double)*nchan*nstokes*nrow, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(dsize_dsigma, data.dsize_dsigma, sizeof(double)*nchan*nstokes*nrow, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(pos_real, data.pos_real, sizeof(double)*nchan*nstokes*nrow, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(pos_imag, data.pos_imag, sizeof(double)*nchan*nstokes*nrow, hipMemcpyDeviceToHost));
}

__global__ void cu_gaussian_size(double sigma, double x0, double y0,
                                 const int nchan, const int nstokes, const int nrow,
                                 double* u, double* v,
                                 double* size)
{
	size_t chan = threadIdx.x;
	size_t pol = threadIdx.y;
	size_t row = blockIdx.x;

	if(chan < nchan and pol < nstokes and row < nrow)
	{
		size_t index = chan+pol*nchan+row*nchan*nstokes;
		size[index] = exp(-2*M_PI*M_PI*(u[index]*u[index]+v[index]*v[index])*sigma*sigma);
	}
}

__global__ void cu_pos(double sigma, double x0, double y0,
                       const int nchan, const int nstokes, const int nrow,
                       double* u, double* v,
                       double* pos_real, double* pos_imag)
{
	size_t chan = threadIdx.x;
	size_t pol = threadIdx.y;
	size_t row = blockIdx.x;

	if(chan < nchan and pol < nstokes and row < nrow)
	{
		size_t index = chan+pol*nchan+row*nchan*nstokes;
		pos_real[index] = cos(-2*M_PI*(x0*u[index]+y0*v[index]));
		pos_imag[index] = sin(-2*M_PI*(x0*u[index]+y0*v[index]));
	}
}


__global__ void cu_evaluate_gaussian(float flux, float sigma, float x0, float y0,
                             const int nchan, const int nstokes, const int nrow,
					         const VisDataContainer uvdata,
							 float* residuals, float* jacobians)
{
	size_t chan = threadIdx.x;
	size_t pol = threadIdx.y;
	size_t row = blockIdx.x;

	if(chan < nchan and pol < nstokes and row < nrow)
	{
		size_t index = chan+pol*nchan+row*nchan*nstokes;

		float& u = uvdata.u[index];
		float& v = uvdata.v[index];
		float& sqrt_weight = uvdata.sqrt_weights[pol+row*nstokes*nchan];

		float size     = exp(-2*M_PI*M_PI*(u*u+v*v)*sigma*sigma);
		float pos_real = cos(-2*M_PI*(x0*u+y0*v));
		float pos_imag = sin(-2*M_PI*(x0*u+y0*v));
		float V_mod_real = flux*size*pos_real;
		float V_mod_imag = flux*size*pos_imag;

		if(uvdata.flag[index])
		{
// 			residuals[2*index+0] = index;
// 			residuals[2*index+1] = index;
			residuals[2*index+0] = 0.0;
			residuals[2*index+1] = 0.0;
			if(jacobians != NULL)
			{
				int nres = 2*nchan*nstokes*nrow;
				jacobians[0*nres+2*index+0] = 0.;
				jacobians[0*nres+2*index+1] = 0.;
				jacobians[1*nres+2*index+0] = 0.;
				jacobians[1*nres+2*index+1] = 0.;
				jacobians[2*nres+2*index+0] = 0.;
				jacobians[2*nres+2*index+1] = 0.;
				jacobians[3*nres+2*index+0] = 0.;
				jacobians[3*nres+2*index+1] = 0.;
			}
		}
		else
		{
// 			residuals[2*index+0] = 1.;
// 			residuals[2*index+1] = 1.;
// 			residuals[2*index+0] = flux;
// 			residuals[2*index+1] = flux;
// 			residuals[2*index+0] = V_mod_real;
// 			residuals[2*index+1] = V_mod_imag;
// 			residuals[2*index+0] = (uvdata.V_real[index]-V_mod_real);
// 			residuals[2*index+1] = (uvdata.V_imag[index]-V_mod_imag);
// 			residuals[2*index+0] = index;
// 			residuals[2*index+1] = index;
			residuals[2*index+0] = sqrt_weight*(uvdata.V_real[index]-V_mod_real);
			residuals[2*index+1] = sqrt_weight*(uvdata.V_imag[index]-V_mod_imag);
			if(jacobians != NULL)
			{
				int nres = 2*nchan*nstokes*nrow;
				jacobians[0*nres+2*index+0] = -sqrt_weight*size*pos_real; // dchi/dflux
				jacobians[0*nres+2*index+1] = -sqrt_weight*size*pos_imag;
				jacobians[1*nres+2*index+0] = -2*M_PI*sqrt_weight*u*V_mod_imag; // dchi/u
				jacobians[1*nres+2*index+1] = 2*M_PI *sqrt_weight*u*V_mod_real;
				jacobians[2*nres+2*index+0] = -2*M_PI*sqrt_weight*v*V_mod_imag; // dchi/v
				jacobians[2*nres+2*index+1] = 2*M_PI *sqrt_weight*v*V_mod_real;
				jacobians[3*nres+2*index+0] = sqrt_weight*V_mod_real * 2*M_PI*M_PI * 2*sigma*(u*u+v*v); // dchi/sigma
				jacobians[3*nres+2*index+1] = sqrt_weight*V_mod_imag * 2*M_PI*M_PI * 2*sigma*(u*u+v*v);
			}
		}
	}
}
__global__ void cu_disk_size(double sigma, double x0, double y0,
                             const int nchan, const int nstokes, const int nrow,
                             double* u, double* v,
                             double* size, double* dsize_dsigma)
{
	size_t chan = threadIdx.x;
	size_t pol = threadIdx.y;
	size_t row = blockIdx.x;

	if(chan < nchan and pol < nstokes and row < nrow)
	{
		size_t index = chan+pol*nchan+row*nchan*nstokes;
		double uvdist = sqrt(u[index]*u[index]+v[index]*v[index]);
		if(sigma*uvdist < 1e-12)
		{
			size[index] = .5;
			dsize_dsigma[index] = 0.;
		}
		else
		{
			size[index] = 1/M_PI*j1(2*M_PI*sigma*uvdist)/sigma/uvdist;
			dsize_dsigma[index] = 2/M_PI*(M_PI*j0(2*M_PI*sigma*uvdist)/sigma -
					j1(2*M_PI*sigma*uvdist)/sigma/sigma/uvdist);
		}
	}
}
