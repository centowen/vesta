#include "hip/hip_runtime.h"
// vesta, use Ceres to douv-model fitting for ms-data.
// Copyright (C) 2015  Lukas Lindroos
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public License
// as published by the Free Software Foundation; either version 2
// of the License, or (at your option) any later version.
// 
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. 
//
#include <iostream>
#include "cuda_error.h"
#include "GaussianCostFunctionCircularCuda.h"

__global__ void cu_gaussian_size(double sigma, double x0, double y0,
                                 const int nchan, const int nstokes, const int nrow,
                                 double* u, double* v,
                                 double* size)
{
	size_t chan = threadIdx.x;
	size_t pol = threadIdx.y;
	size_t row = blockIdx.x;

	if(chan < nchan and pol < nstokes and row < nrow)
	{
		size_t index = chan+pol*nchan+row*nchan*nstokes;
		size[index] = exp(-2*M_PI*M_PI*(u[index]*u[index]+v[index]*v[index])*sigma*sigma);
	}
};

__global__ void cu_pos(double sigma, double x0, double y0,
                       const int nchan, const int nstokes, const int nrow,
                       double* u, double* v,
                       double* pos_real, double* pos_imag)
{
	size_t chan = threadIdx.x;
	size_t pol = threadIdx.y;
	size_t row = blockIdx.x;

	if(chan < nchan and pol < nstokes and row < nrow)
	{
		size_t index = chan+pol*nchan+row*nchan*nstokes;
		pos_real[index] = cos(-2*M_PI*(x0*u[index]+y0*v[index]));
		pos_imag[index] = sin(-2*M_PI*(x0*u[index]+y0*v[index]));
	}
};

void calc_functions(double sigma, double x0, double y0,
                    const int nchan, const int nstokes, const int nrow,
                    double* u, double* v,
                    double* size, double* pos_real, double* pos_imag,
					const DataContainer data)
{
	CudaSafeCall(hipMemcpy( data.u, u, sizeof(double)*nchan*nstokes*nrow, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy( data.v, v, sizeof(double)*nchan*nstokes*nrow, hipMemcpyHostToDevice));
	dim3 dimBlock(nchan, nstokes);
	dim3 dimGrid(nrow);
	cu_gaussian_size<<<dimGrid, dimBlock>>>(sigma, x0, y0, nchan, nstokes, nrow,
	                                        data.u, data.v, data.size);
	cu_pos<<<dimGrid, dimBlock>>>(sigma, x0, y0, nchan, nstokes, nrow,
	                              data.u, data.v, data.pos_real, data.pos_imag);

	CudaSafeCall(hipMemcpy(size, data.size, sizeof(double)*nchan*nstokes*nrow, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(pos_real, data.pos_real, sizeof(double)*nchan*nstokes*nrow, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(pos_imag, data.pos_imag, sizeof(double)*nchan*nstokes*nrow, hipMemcpyDeviceToHost));
}

void allocate_stuff(const int nchan, const int nstokes, const int nrow, DataContainer& data)
{
	CudaSafeCall(hipMalloc( (void**)&data.u, sizeof(double)*nchan*nstokes*nrow));
	CudaSafeCall(hipMalloc( (void**)&data.v, sizeof(double)*nchan*nstokes*nrow));
	CudaSafeCall(hipMalloc( (void**)&data.size, sizeof(double)*nchan*nstokes*nrow));
	CudaSafeCall(hipMalloc( (void**)&data.pos_real, sizeof(double)*nchan*nstokes*nrow));
	CudaSafeCall(hipMalloc( (void**)&data.pos_imag, sizeof(double)*nchan*nstokes*nrow));
}

void free_stuff(DataContainer& data)
{
	CudaSafeCall(hipFree( data.u));
	CudaSafeCall(hipFree( data.v));
	CudaSafeCall(hipFree( data.size));
	CudaSafeCall(hipFree( data.pos_real));
	CudaSafeCall(hipFree( data.pos_imag));
}
