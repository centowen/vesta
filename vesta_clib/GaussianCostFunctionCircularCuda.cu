#include "hip/hip_runtime.h"
// vesta, use Ceres to douv-model fitting for ms-data.
// Copyright (C) 2015  Lukas Lindroos
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public License
// as published by the Free Software Foundation; either version 2
// of the License, or (at your option) any later version.
// 
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. 
//
#include <iostream>
#include "cuda_error.h"

double* dev_u;
double* dev_v;
double* dev_size;
double* dev_pos_real;
double* dev_pos_imag;

__global__ void cuEvaluate(double sigma, double x0, double y0, int nchan, int nstokes,
                           double* u, double* v,
                           double* size, double* pos_real, double* pos_imag)
{
	size_t chan = threadIdx.x;
	size_t pol = blockIdx.x;

	if(chan < nchan and pol < nstokes)
	{
		size_t index = chan+pol*nchan;
		size[index] = exp(-2*M_PI*M_PI*(u[index]*u[index]+v[index]*v[index])*sigma*sigma);
		pos_real[index] = cos(-2*M_PI*(x0*u[index]+y0*v[index]));
		pos_imag[index] = sin(-2*M_PI*(x0*u[index]+y0*v[index]));
	}
};

void calc_functions(double sigma, double x0, double y0, int nchan, int nstokes,
                    double* u, double* v,
					double* size, double* pos_real, double* pos_imag)
{
	CudaSafeCall(hipMemcpy( dev_u, u, sizeof(double)*nchan*nstokes, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy( dev_v, v, sizeof(double)*nchan*nstokes, hipMemcpyHostToDevice));
	cuEvaluate<<<nchan, nstokes>>>(sigma, x0, y0, nchan, nstokes, dev_u, dev_v, dev_size, dev_pos_real, dev_pos_imag);
	CudaSafeCall(hipMemcpy(size, dev_size, sizeof(double)*nchan*nstokes, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(pos_real, dev_pos_real, sizeof(double)*nchan*nstokes, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(pos_imag, dev_pos_imag, sizeof(double)*nchan*nstokes, hipMemcpyDeviceToHost));
}

void allocate_stuff(const int nchan, const int nstokes)
{
	CudaSafeCall(hipMalloc( (void**)&dev_u, sizeof(double)*nchan*nstokes));
	CudaSafeCall(hipMalloc( (void**)&dev_v, sizeof(double)*nchan*nstokes));
	CudaSafeCall(hipMalloc( (void**)&dev_size, sizeof(double)*nchan*nstokes));
	CudaSafeCall(hipMalloc( (void**)&dev_pos_real, sizeof(double)*nchan*nstokes));
	CudaSafeCall(hipMalloc( (void**)&dev_pos_imag, sizeof(double)*nchan*nstokes));
}

void free_stuff()
{
	CudaSafeCall(hipFree( dev_u));
	CudaSafeCall(hipFree( dev_v));
	CudaSafeCall(hipFree( dev_size));
	CudaSafeCall(hipFree( dev_pos_real));
	CudaSafeCall(hipFree( dev_pos_imag));
}
